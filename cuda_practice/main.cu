// Multiblock MatMul with flattened Matrices.
#include <stdio.h>
#include <hip/hip_runtime.h>

// Let's first practice doing it on CPU
__host__ void matmul_kernel_cpu(float* Md, float* Nd, float* Pd, int Width){
    // Assume same dims and no threads for a moment
    // Width is therefore one side (row or col) of the matrices, all of same width
    // But assume flattened matrices
    for (int i = 0; i < Width; i++){
        // iterate through the rows vectors of the first matrix
        for (int j = 0; j < Width; j++){
            // iterate through columns of the second matrix
            float dp = 0.0f;
            for (int k = 0; k < Width; k++){
                // iterate through the elements of each vector in both matrices
                float r = Md[i*Width + k]; // row-wise for the first matrix
                float c = Nd[k*Width + j]; // column-wise for the second matrix
                dp += r*c; // summate the products
            }
            // place dot product in result matrix
            Pd[i*Width + j] = dp; // at the ith row and jth col.
        }
    }

    // Print the resulting matrix Pd from within the kernel.
    printf("\nResult Matrix Pd (from kernel):\n");
    for(int i = 0; i < Width; i++) {
        for(int j = 0; j < Width; j++) {
            printf("%.1f ", Pd[i * Width + j]);
        }
        printf("\n");
    }
}

__global__ void matmul_kernel_gpu_singleblock(float* Md, float* Nd, float* Pd, int Width){
    // This will be one of the threads in the thread block -- specifically at the following
    // These correspond to the row / column in the Md and Nd matrices
    int i = threadIdx.x; // This is the col in Pd (and therefore also in Nd)
    int j = threadIdx.y; // This is the row in Pd (and therefore also in Md)
    float dp = 0.0f;
    for (int k = 0; k < Width; k++){ // k remains the column in Md and row in Nd
        // Applying index = row*width + col
        float r = Md[j*Width + k]; // row-wise for the first matrix
        float c = Nd[k*Width + i]; // column-wise for the second matrix
        dp += r*c; // summate the products
    }
    Pd[j*Width + i] = dp; // You can't just append I guess? so you apply row*width + col
}

__global__ void matmul_kernel_gpu_multiblock(float* Md, float* Nd, float* Pd, int Width){
    // Don't assume any specific Width, but assume it's greater than BlockDim.x == BlockDim.y --> Multiple blocks == (GridDim =/= 1x1)
    // --> Tiling --> Bounds checking! 
    int row = blockIdx.y*blockDim.y + threadIdx.y; // Global Thread ID, which can be used for indexing. Specifically, this is the row in Pd (and therefore also in Md).
    int col = blockIdx.x*blockDim.x + threadIdx.x; // Global Thread ID, which can be used for indexing. Specifically, this is the col in Pd (and therefore also in Nd).
    if (row < Width && col < Width){ // Skip if out of bounds
        float dp = 0.0f;
        for (int k = 0; k < Width; k++){ // For indexing within the vectors. This is a col in Md and a row in Nd. 
            // indexing in flattened matrix: flat_index = row_of_element_in_OG_Matrix*width_of_Matrix + col_of_element_in_OG_matrix (!) i.e., index = row*width + col
            dp += Md[row*Width + k] * Nd[k*Width + col];
        }
        // Again, same indexing strategy. If you name every variable appropriately and understand what it represents, this becomes clear.
        Pd[row*Width + col] = dp;
    }
}

void matmul(float* M, float* N, float* P, int Width){
    // Create pointers for new memory on device and allocate the new memory 
    float* Md;
    int sizeM = Width*Width*sizeof(float);
    hipMalloc((void**)&Md, sizeM);
    float* Nd;
    int sizeN = Width*Width*sizeof(float);
    hipMalloc((void**)&Nd, sizeN);
    float* Pd;
    int sizeP = Width*Width*sizeof(float);
    hipMalloc((void**)&Pd, sizeP);
    printf("Memory allocated.\n");
    // Copy the data to the new memory addresses given to the pointers by cudaMalloc()
    hipMemcpy(Md, M, sizeM, hipMemcpyHostToDevice);
    hipMemcpy(Nd, N, sizeN, hipMemcpyHostToDevice);
    // Do the MatMul
    // matmul_kernel_cpu(M, N, P, Width); // CPU version
    dim3 dimBlock(2, 2);
    dim3 dimGrid(1, 1);
    matmul_kernel_gpu_multiblock<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width); // GPU version
    // Free the previously allocated device memory
    hipMemcpy(P, Pd, sizeP, hipMemcpyDeviceToHost);
    // Print the matrix P from the host right after copying it back from the device.
    printf("\nResult Matrix P (from matmul, after GPU computation):\n");
    for(int i = 0; i < Width; i++) {
        for(int j = 0; j < Width; j++) {
            printf("%.1f ", P[i * Width + j]);
        }
        printf("\n");
    }

    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
    printf("Memory freed.\n");
}

int main(){
    float M[4] = {1, 2, 3, 4};
    float N[4] = {5, 6, 7, 8};
    float P[4];
    matmul(M, N, P, 2); // M is converted to &M[0]
}